#include "hip/hip_runtime.h"
/*
 * ---------------------------------- LICENSE ----------------------------------
 * This software is in the public domain.  Where that dedication is not
 * recognized, you are granted a perpetual, irrevocable license to copy,
 * distribute, and modify the source code as you see fit.
 *
 * The source code is provided "as is", without warranty of any kind, express
 * or implied. No attribution is required, but always appreciated.
 * =============================================================================
 *
 */

#include <stdio.h>
#include <stdlib.h>

#if defined(_WIN32) || defined(_WIN64)
#include <windows.h>
#endif

#include <hip/hip_runtime.h>
#include <math_functions.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <hip/hip_vector_types.h>

#include <stdint.h>
typedef int32_t i32;
typedef uint32_t u32;
typedef uint64_t u64;
typedef size_t usize;
typedef int32_t b32;

#include <float.h>

typedef float f32;

#define F32_MAX FLT_MAX

#define HALF_PI32 1.570796327f

typedef struct {
    float3 pos;
    float3 dir;
} rtRay;

typedef struct {
    float3 pos;
    f32 r;
    float3 diffuse;
} rtSphere;

typedef struct {
    float3 pos;
    float3 emiss;
} rtLight;

#define TEX_PITCH 4
#define SAMPLE_COUNT 8
#define R_SAMPLE_COUNT 1.0f / (f32) SAMPLE_COUNT
#define BLOCK_COUNT 8

#define LIGHT_COUNT 1
static rtLight h_light[LIGHT_COUNT] = {
    { make_float3(0.0f, -15.0f, 0.0f), make_float3(1.0f) },
};

#define SPHERE_COUNT 6
static rtSphere h_spheres[SPHERE_COUNT] = {
    { make_float3(0.0f, 0.0f, 0.0f), 1.0f, make_float3(0.5f) },
    { make_float3(0.0f, 6.0f, 0.0f), 4.0f, make_float3(1.0f, 0.0f, 0.0f) },
    { make_float3(0.0f, -6.0f, 0.0f), 3.0f, make_float3(1.0f) },
    { make_float3(6.0f, 0.0f, -5.0f), 3.0f, make_float3(0.0f, 1.0f, 0.0f) },
    { make_float3(-6.0f, 0.0f, 0.0f), 3.0f, make_float3(0.0f, 0.0f, 1.0f) },
    { make_float3(0.0f, 10010.0f, 0.0f), 10000.0f, make_float3(0.75f) },
};

static u32 gl_tex;
static hipGraphicsResource* d_resource = NULL;
__constant__ rtSphere d_spheres[SPHERE_COUNT];
__constant__ rtLight d_light[LIGHT_COUNT];

__device__ inline void ray_project(rtRay* ray, u32 px, u32 py, i32 width, i32 height, f32 fov) {
    f32 ar = __fdividef((f32) width, (f32) height);
    f32 a = __tanf(HALF_PI32 * fov / 180.0f);
    f32 xx = (2.0f * ((px + 0.5f) * __fdividef(1.0f, (f32) width)) - 1) * a * ar;
    f32 yy = (1.0f - 2.0f * ((py + 0.5f) * __fdividef(1.0f, (f32) height))) * a;
    ray->pos = make_float3(0.0f, 0.0f, 20.0f);
    ray->dir = normalize(make_float3(xx, yy, -1.0f));
}

__device__ inline b32 ray_sphere_intersect(rtRay* ray, rtSphere* s, f32* dist) {
    float3 light_dir = s->pos - ray->pos;
    f32 sz = fmaxf(0.0f, dot(light_dir, ray->dir));
    f32 dsq = dot(light_dir, light_dir) - __powf(sz, 2);
    f32 rsq = __powf(s->r, 2);
    f32 q = __fsqrt_rn(rsq - dsq);
    *dist = sz - q;
    return dsq < rsq;
}

__global__ void raytrace_kernel(hipSurfaceObject_t surf, i32 width, i32 height, f32 fov) {
    const u32 x = blockIdx.x * blockDim.x + threadIdx.x;
    const u32 y = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ float3 pixels[BLOCK_COUNT][BLOCK_COUNT];
    pixels[threadIdx.x][threadIdx.y] = make_float3(0.0f);

    __shared__ rtRay rays[BLOCK_COUNT][BLOCK_COUNT];
    ray_project(&rays[threadIdx.x][threadIdx.y], x, y, width, height, fov);

    f32 tn = F32_MAX;
    rtSphere* s = NULL;
#pragma unroll SPHERE_COUNT
    for(i32 i = 0; i < SPHERE_COUNT; i++) {
        f32 dist = F32_MAX;
        if(ray_sphere_intersect(&rays[threadIdx.x][threadIdx.y], &d_spheres[i], &dist)) {
            if(dist < tn) {
                tn = dist;
                s = &d_spheres[i];
            }
        }
    }

    __syncthreads();
    if(s) {
        float3 phit = rays[threadIdx.x][threadIdx.y].pos + rays[threadIdx.x][threadIdx.y].dir * tn;
        float3 nhit = normalize(phit - s->pos);
#pragma unroll LIGHT_COUNT
        for(u32 i = 0; i < LIGHT_COUNT; i++) {
            float3 tr = make_float3(1.0f);
#pragma unroll SAMPLE_COUNT
            for(u32 sa = 0; sa < SAMPLE_COUNT; sa++) {
                float3 ld = normalize(d_light[i].pos + __fdividef(sa, (f32) SAMPLE_COUNT) - phit);
                rtRay light_ray = { phit + nhit * 0.01f, ld };
#pragma unroll SPHERE_COUNT
                for(u32 j = 0; j < SPHERE_COUNT; j++) {
                    f32 dist;
                    if(ray_sphere_intersect(&light_ray, &d_spheres[j], &dist)) {
                        tr = make_float3(0.1f);
                        break;
                    }
                }
                pixels[threadIdx.x][threadIdx.y] += (s->diffuse * tr * fmaxf(0.0f, dot(nhit, ld)) * d_light[i].emiss) * R_SAMPLE_COUNT;
            }
        }
        pixels[threadIdx.x][threadIdx.y] *= s->diffuse;
    }

    pixels[threadIdx.x][threadIdx.y] = clamp(pixels[threadIdx.x][threadIdx.y] * 255.0f, 0.0f, 255.0f);
    __syncthreads();
    surf2Dwrite(
            (0xFF000000 |
             (((u32) pixels[threadIdx.x][threadIdx.y].x) << 16) |
             (((u32) pixels[threadIdx.x][threadIdx.y].y) << 8) |
             (((u32) pixels[threadIdx.x][threadIdx.y].z) << 0)),
            surf, x * TEX_PITCH, y);
}

extern "C" void raytrace_init(i32 width, i32 height) {
    glEnable(GL_TEXTURE_2D);

    glGenTextures(1, &gl_tex);

    glBindTexture(GL_TEXTURE_2D, gl_tex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glBindTexture(GL_TEXTURE_2D, 0);

    hipGraphicsGLRegisterImage(&d_resource, gl_tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);

    hipMemcpyToSymbol(HIP_SYMBOL(d_spheres), h_spheres, SPHERE_COUNT * sizeof(rtSphere));
    hipMemcpyToSymbol(HIP_SYMBOL(d_light), h_light, LIGHT_COUNT * sizeof(rtLight));

    hipProfilerStart();
}

extern "C" void raytrace_destroy() {
    hipProfilerStop();
}

extern "C" void raytrace_resize(i32 width, i32 height) {
    glViewport(0, 0, width, height);
    hipGraphicsUnregisterResource(d_resource);
    glBindTexture(GL_TEXTURE_2D, gl_tex);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glBindTexture(GL_TEXTURE_2D, 0);
    hipGraphicsGLRegisterImage(&d_resource, gl_tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
}

extern "C" void raytrace_render(i32 width, i32 height, f32 t, f32 dt, f32 fov) {
    dim3 block = dim3(BLOCK_COUNT, BLOCK_COUNT, 1);
    dim3 grid = dim3(width / block.x, height / block.y, 1);

    hipGraphicsMapResources(1, &d_resource);
    hipArray_t array;
    hipGraphicsSubResourceGetMappedArray(&array, d_resource, 0, 0);
    hipResourceDesc desc;
    desc.resType = hipResourceTypeArray;
    desc.res.array.array = array;
    hipSurfaceObject_t surf;
    hipCreateSurfaceObject(&surf, &desc);

    h_light[0].pos.x = sinf(t) * 20.0f;
    h_light[0].pos.z = cosf(t) * -20.0f;
    hipMemcpyToSymbol(HIP_SYMBOL(d_light), h_light, LIGHT_COUNT * sizeof(rtLight));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    raytrace_kernel<<<grid, block>>>(surf, width, height, fov);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    f32 ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    hipDestroySurfaceObject(surf);
    hipGraphicsUnmapResources(1, &d_resource);
    hipDeviceSynchronize();

    glEnable(GL_TEXTURE_2D);
    glBindTexture(GL_TEXTURE_2D, gl_tex);

    glBegin(GL_QUADS); {
        glTexCoord2f(0.0f, 0.0f);
        glVertex2f(-1.0f, -1.0f);
        glTexCoord2f(1.0f, 0.0f);
        glVertex2f(1.0f, -1.0f);
        glTexCoord2f(1.0f, 1.0f);
        glVertex2f(1.0f, 1.0f);
        glTexCoord2f(0.0f, 1.0f);
        glVertex2f(-1.0f, 1.0f);
    } glEnd();

    glBindTexture(GL_TEXTURE_2D, 0);
    glDisable(GL_TEXTURE_2D);
}
